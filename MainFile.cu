#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>
#include <time.h>
#include "win-gettimeofday.h"


#define min(a,b) (a<b?a:b)
#define sum_squares(x) ((x*(x+1)*(2*x+1))/6)

const int k = 100;
const int N = k * 1024; //set data size
const int threadsPerBlock = 32;
const int blocksPerGrid = 1;

__device__ void doMerge(int A[], int start, int middle, int end, int B[]) {
	int i = start, j = middle;

	for (int k = start; k < end; k++) {
		if (i < middle && (j >= end || A[i] <= A[j])) {
			B[k] = A[i];
			i++;
		}
		else {
			B[k] = A[j];
			j++;
		}
	}
}

__device__ void doSplitMerge(int B[], int start, int end, int A[]) {
	if (end - start < 2) {
		return;
	}

	int middle = (end + start) / 2;

	doSplitMerge(A, start, middle, B);
	doSplitMerge(A, middle, end, B);

	doMerge(B, start, middle, end, A);
}

__global__ void mergeSort(int *a, int *b, int *elements, int *threads) {
	int block_id = blockIdx.x + gridDim.x * blockIdx.y;
	int thread_id = blockDim.x * block_id + threadIdx.x;  //calculate thread id

	int numberOfElements = N / (blocksPerGrid*threads[0]); //calculate how many elements this process must sort

	elements[thread_id] = numberOfElements;
	
	int startPoint = 0;
	for (int i = 0; i < thread_id; i++){
		startPoint += elements[i]; //calculate where this process must start in the array
	}

	doSplitMerge(b, startPoint, startPoint + numberOfElements, a); //sort the elements

	__syncthreads();
	
	for (int i = startPoint; i < startPoint + numberOfElements; i++){
		b[startPoint + i] = a[startPoint + i]; //make the backup array equal the uptodate array
	}
	__syncthreads();
}

int main(void) {
	int *a,
	int *dev_a,
	int *b,
	int *dev_b;
	int *cputhreads;
	int *elementsPerThread;
	int *threads;

	// allocate memory on the CPU
	a = (int*)malloc(N * sizeof(int));
	b = (int*)malloc(N * sizeof(int));
	cputhreads = (int*)malloc(sizeof(int));

	cputhreads[0] = threadsPerBlock; //set number of threads to constant variable
	
	// allocate memory on the GPU
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&elementsPerThread, (threadsPerBlock * blocksPerGrid) * sizeof(int));
	hipMalloc((void**)&threads,sizeof(int));

	// fill in the host memory with data
	for (int i = 0; i<N; i++) {
		a[i] = rand() % 1000;
		b[i] = a[i];
	}
	
	// start timer
	long long memory_start_time = start_timer();

	// copy the arrays a and b to the GPU
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	for (int i = cputhreads[0]; i > 0; i /= 2) {
		cputhreads[0] = i;
		hipMemcpy(threads, cputhreads, sizeof(int), hipMemcpyHostToDevice); //update GPU with number of threads
		mergeSort << <blocksPerGrid, cputhreads[0] >> > (dev_a, dev_b, elementsPerThread, threads);
		hipDeviceSynchronize(); // make sure all threads have finished processing
	}

	// copy the array �a� back from the GPU to the CPU
	hipMemcpy(a, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);

	// stop timer and print result
	stop_timer(memory_start_time, "\nGPU:\t Compute Sort");
			
	/* print the data */
	printf("Sorted array ");
	for (int i = 0; i < N; i++) {
		printf("%d, ", a[i]);
	}
	// free memory on the GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(elementsPerThread);
	hipFree(threads);
	// free memory on the CPU
	free(a);
	free(b);
	free(cputhreads);

	getchar();
	return 0;
}